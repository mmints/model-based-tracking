#include "hip/hip_runtime.h"
#include <cstdio>
#include "General.h"

#define N 100000 // vector size

__global__
void addKernel(int *a, int *b, int *c)
{
    // calculate the index position in grid
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < N)
    {
        c[index] = a[index] + b[index];
    }
}

void kernel::add()
{
    int a[N], b[N], c[N];       // voctors of size N
    int *dev_a, *dev_b, *dev_c; // pointer to device memory space

    // Allocate memory space on gpu
    hipMalloc((void**) &dev_a, N * sizeof(int));
    hipMalloc((void**) &dev_b, N * sizeof(int));
    hipMalloc((void**) &dev_c, N * sizeof(int));

    // Just for getting some numbers
    // Fill vectors on CPU
    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = i * i;
    }

    // copy vectors from CPU to GPU memory
    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice); // a is the pointer to a[0]
    hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, N * sizeof(int), hipMemcpyHostToDevice);

    // call the kernel with as much as possible block with at least 128 threads each (utilize modulo for integer)
    addKernel<<<(N + 127)/128, 128>>>(dev_a, dev_b, dev_c);

    // copy result back from GPU to CPU memory
    hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

    // print out the results
    for (int i = 0; i < N; i++) {
        printf("%d + %d = %d \n", a[i], b[i], c[i]);
    }

    // clear memory on gpu
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}