#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <>
#include "kernel.h"
#include <ErrorHandling/HANDLE_CUDA_ERROR.h>


texture<uchar4, 2, hipReadModeElementType> tex_ref;

__global__ void kernel(sl::uchar4 *zed_in, sl::uchar4 *zed_out,  size_t step) {
    uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
    uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;
    uint32_t offset = x + y * step;

    uchar4 texel_value = tex2D(tex_ref, x, y);

    if (texel_value.x == 0 && texel_value.y == 0 && texel_value.z == 0) {
        zed_out[offset].x = zed_in[offset].z;
        zed_out[offset].y = zed_in[offset].y;
        zed_out[offset].z = zed_in[offset].x;
        return;
    }
    zed_out[offset].x = texel_value.x;
    zed_out[offset].y = texel_value.y;
    zed_out[offset].z = texel_value.z;
}

void callKernel(sl::uchar4 *zed_in, sl::uchar4 *zed_out,  size_t step, int width, int height, hipArray *tex_array)
{
/*
    hipChannelFormatDesc desc;
    HANDLE_CUDA_ERROR(hipGetChannelDesc(&desc, tex_array));

    printf("++++ Size of texture: %i \n", sizeof(uchar4));

    printf("CUDA Array channel descriptor, bits per component:\n");
    printf("X %d Y %d Z %d W %d, kind %d\n",
           desc.x,desc.y,desc.z,desc.w,desc.f);
*/

    HANDLE_CUDA_ERROR(hipBindTextureToArray(tex_ref, tex_array));

    const size_t BLOCKSIZE_X = 32;
    const size_t BLOCKSIZE_Y = 8;

    dim3 dimBlock{BLOCKSIZE_X,BLOCKSIZE_Y};
    dim3 dimGrid;

    dimGrid.x = (width + dimBlock.x - 1) / dimBlock.x;
    dimGrid.y = (height + dimBlock.y - 1) / dimBlock.y;

    kernel<<<dimGrid, dimBlock>>>(zed_in, zed_out, step);
    HANDLE_CUDA_ERROR(hipUnbindTexture(tex_ref));

}

__global__ void kernel2() {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    uchar4 texel_value = tex2D(tex_ref, x, y);

    if (texel_value.x != 0 || texel_value.y != 0 || texel_value.z != 0) {
        printf("(%i, %i) = r: %i, g: %i, b %i \n",x, y, texel_value.x, texel_value.y, texel_value.z);
    }
}

void callKernel2(int width, int height, hipArray *tex_array)
{
    hipChannelFormatDesc desc;
    HANDLE_CUDA_ERROR(hipGetChannelDesc(&desc, tex_array));

    printf("++++ Size of texture: %i \n", sizeof(uchar4));

    printf("CUDA Array channel descriptor, bits per component:\n");
    printf("X %d Y %d Z %d W %d, kind %d\n",
           desc.x,desc.y,desc.z,desc.w,desc.f);


    HANDLE_CUDA_ERROR(hipBindTextureToArray(tex_ref, tex_array));

    const size_t BLOCKSIZE_X = 32;
    const size_t BLOCKSIZE_Y = 8;

    dim3 dimBlock{BLOCKSIZE_X,BLOCKSIZE_Y};
    dim3 dimGrid;

    dimGrid.x = (width + dimBlock.x - 1) / dimBlock.x;
    dimGrid.y = (height + dimBlock.y - 1) / dimBlock.y;

    kernel2<<<dimGrid, dimBlock>>>();
    HANDLE_CUDA_ERROR(hipUnbindTexture(tex_ref));

}

