#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <>
#include "kernel.h"
#include <ErrorHandling/HANDLE_CUDA_ERROR.h>


texture<float4 , 2, hipReadModeElementType> tex_ref;

__global__ void kernel(sl::uchar1 *d_in, sl::uchar1 *d_out, size_t step) {
    uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
    uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;

    // if texture-pixel-color != 0,0,0
        // write d_in-pixel into d_out-pixel
    // else:
        // write texture-pixel into d_out-pixel

}

void callKernel(int width, int height, hipArray *tex_array, sl::uchar1 *d_in, sl::uchar1 *d_out, size_t step)
{
    hipChannelFormatDesc desc;
    HANDLE_CUDA_ERROR(hipGetChannelDesc(&desc, tex_array));

    printf("CUDA Array channel descriptor, bits per component:\n");
    printf("X %d Y %d Z %d W %d, kind %d\n",
           desc.x,desc.y,desc.z,desc.w,desc.f);

    HANDLE_CUDA_ERROR(hipBindTextureToArray(tex_ref, tex_array));

    const size_t BLOCKSIZE_X = 32;
    const size_t BLOCKSIZE_Y = 8;

    dim3 dimBlock{BLOCKSIZE_X,BLOCKSIZE_Y};
    dim3 dimGrid;

    dimGrid.x = (width + dimBlock.x - 1) / dimBlock.x;
    dimGrid.y = (height + dimBlock.y - 1) / dimBlock.y;

    kernel<<<dimGrid, dimBlock>>>(d_in, d_out, step);
}


