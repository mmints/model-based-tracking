#include "hip/hip_runtime.h"
#include "ModelTracker/kernel_functions.h"
#include <cstdlib>

texture<uchar4, 2, hipReadModeElementType> particle_grid_texture_ref;

__device__ void compare(float &weight, const uchar4 &particle_pixel, const sl::uchar4 &zed_pixel, const int threshold)
{
        int diff_x = std::abs(particle_pixel.x - zed_pixel.x);
        int diff_y = std::abs(particle_pixel.y - zed_pixel.y);
        int diff_z = std::abs(particle_pixel.z - zed_pixel.z);

        if (diff_x < threshold && diff_y < threshold && diff_z < threshold)
        {
            weight = 1.f;
        }
        else {
            weight = 0.f;
        }
}


__global__ void calculateWeightKernel(sl::uchar4 *zed_in, size_t step, int particle_scale,
                                      int particle_grid_dimension, int particle_width, int particle_height,
                                      float *weight_memory)
{
    // Get the pixel value from particleGrid.texture (parts as particle_grid_texture_ref)
    // use unsigned integer because the numbers can become very large
    uint32_t particle_grid_texture_x = threadIdx.x + blockIdx.x * blockDim.x;
    uint32_t particle_grid_texture_y = threadIdx.y + blockIdx.y * blockDim.y;
    uchar4 particle_grid_pixel_value = tex2D(particle_grid_texture_ref, particle_grid_texture_x, particle_grid_texture_y);

    // Transfer particle grid pixel coordinate to ZED pixel coordinate
    uint32_t zed_x = (particle_grid_texture_x % particle_width) * particle_scale;
    uint32_t zed_y = (particle_grid_texture_y % particle_height) * particle_scale;
    uint32_t offset = zed_x + zed_y * step; // Flat coordinate to memory space

    // Calculate the index of the current corresponding particle to the given texel
    int particle_index = (int)(particle_grid_texture_x / particle_width) + (int)(particle_grid_texture_y / particle_height) * particle_grid_dimension;

    // Calculate weight per pixel
    float weight = 0.f;

    // Give every pixel the weight 1, so the sum should be particle_width x particle_height
    weight = 1.f;
    atomicAdd(&weight_memory[particle_index], weight);


/*    // Fill the weight array with particle indexes for testing
    weight_memory[particle_index] = (float) particle_index;*/
}

void mt::calculateWeight(const sl::Mat &in_zed, float *dev_weight_memory, hipArray *particle_grid_tex_array, mt::ParticleGrid &particleGrid)
{
    HANDLE_CUDA_ERROR(hipBindTextureToArray(particle_grid_texture_ref, particle_grid_tex_array));

    size_t width = in_zed.getWidth();
    size_t height = in_zed.getHeight();
    size_t step = in_zed.getStep(sl::MEM_GPU);

    int particle_width = particleGrid.getParticleWidth();
    int particle_height = particleGrid.getParticleHeight();
    int particle_gird_dimension = particleGrid.getParticleGridDimension();

    int particle_scale = width / particle_width;
    sl::uchar4 *in_zed_ptr = in_zed.getPtr<sl::uchar4>(sl::MEM_GPU);

    const size_t BLOCKSIZE_X = 32;
    const size_t BLOCKSIZE_Y = 8;

    dim3 dimBlock{BLOCKSIZE_X,BLOCKSIZE_Y};
    dim3 dimGrid;

    dimGrid.x = (particle_width * particle_gird_dimension + dimBlock.x - 1) / dimBlock.x;
    dimGrid.y = (particle_height * particle_gird_dimension + dimBlock.y - 1) / dimBlock.y;

    calculateWeightKernel<<<dimGrid, dimBlock>>>(in_zed_ptr, step, particle_scale,
            particle_gird_dimension, particle_width, particle_height,
            dev_weight_memory);

    HANDLE_CUDA_ERROR(hipUnbindTexture(particle_grid_texture_ref));
}
