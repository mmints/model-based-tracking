#include "hip/hip_runtime.h"
#include <cstdint>

#include "ImageFilter.h"

__global__ void simpleRedDetectorKernel(sl::uchar4 *d_in, sl::uchar4 *d_out, size_t step, unsigned char threshold) {

    // Flat the 2D Coordinates to 1D
    uint32_t zed_x = threadIdx.x + blockIdx.x * blockDim.x;
    uint32_t zed_y = threadIdx.y + blockIdx.y * blockDim.y;
    uint32_t offset = zed_x + zed_y * step;

    sl::uchar4 pixel_color = d_in[offset];
    sl::uchar4 red = sl::uchar4(255, 0, 0, 0);

    // The values from ZED are coming in BGR
    if (pixel_color.z >= threshold && pixel_color.y < 100 && pixel_color.x < 100) {
        d_out[offset] = red;
    }
    else {
        d_out[offset] = sl::uchar4(0, 0, 0, 0);
    }
}

// Returns a red color map in d_out
void kernel::simpleRedDetector(sl::uchar4 *d_in, sl::uchar4 *d_out, unsigned char threshold, int width, int height, size_t step)
{
    const size_t BLOCKSIZE_X = 32;
    const size_t BLOCKSIZE_Y = 8;

    dim3 dimBlock{BLOCKSIZE_X,BLOCKSIZE_Y};
    dim3 dimGrid;

    dimGrid.x = (width + dimBlock.x - 1) / dimBlock.x;
    dimGrid.y = (height + dimBlock.y - 1) / dimBlock.y;

    // TODO: Add HSV transformation kernel to remove lightning artifacts
    simpleRedDetectorKernel<<<dimGrid, dimBlock>>>(d_in, d_out, step, threshold);
}